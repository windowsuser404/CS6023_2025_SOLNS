/**
 *   CS6023: GPU Programming
 *   Assignment 1
 *
 *   Please don't change any existing code in this file.
 *
 *   You can add your code whereever needed. Please add necessary memory APIs
 *   for your implementation. Use cudaFree() to free up memory as soon as you're
 *   done with an allocation. This will ensure that you don't run out of memory
 *   while running large test cases. Use the minimum required memory for your
 *   implementation. DO NOT change the kernel configuration parameters.
 */

#include <chrono>
#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <math.h>

using std::cin;
using std::cout;

typedef long int lint;
__global__ void CalculateInvertedGrayScale(lint *d_red, lint *d_green,
                                           lint *d_blue, lint *Out, int rows,
                                           int cols) {
  // calculate you id
  lint total_thread_per_blk = blockDim.x * blockDim.y * blockDim.z;
  lint blk_id =
      gridDim.y * gridDim.x * blockIdx.z + gridDim.x * blockIdx.y + blockIdx.x;
  lint uniq_id = threadIdx.x + threadIdx.y * blockDim.x +
                 threadIdx.z * blockDim.x * blockDim.y +
                 blk_id * total_thread_per_blk;
  // calculate the co-ordunates to write at
  lint x = uniq_id % cols;
  lint y = uniq_id / cols;
  if (y < rows) {
    lint grey =
        d_red[x + y * cols] + d_blue[x + y * cols] + d_green[x + y * cols];
    grey = grey / 3;
    Out[x + (rows - 1 - y) * cols] = grey;
  }
}

__global__ void CalculateThomasTransformation(lint *d_red, lint *d_green,
                                              lint *d_blue, lint *Out, int rows,
                                              int cols) {
  // calculate you id
  lint total_thread_per_blk = blockDim.x * blockDim.y * blockDim.z;
  lint blk_id =
      gridDim.y * gridDim.x * blockIdx.z + gridDim.x * blockIdx.y + blockIdx.x;
  lint uniq_id = threadIdx.x + threadIdx.y * blockDim.x +
                 threadIdx.z * blockDim.x * blockDim.y +
                 blk_id * total_thread_per_blk;
  // calculate the co-ordunates to write at
  lint x = uniq_id % cols;
  lint y = uniq_id / cols;
  if (y < rows) {
    lint grey = floor(0.5 * d_red[x + y * cols]) + d_blue[x + y * cols] +
                floor(sqrt((float)d_green[x + y * cols]));
    Out[x + (y)*cols] = grey;
  }
}

int main(int argc, char **argv) {
  const char *inputFileName = argv[1];

  FILE *inputFile = NULL;

  // Open the file for reading
  if ((inputFile = fopen(inputFileName, "r")) == NULL) {
    printf("Failed at opening the file %s\n", inputFileName);
    return 1; // Exit with an error code
  }
  int m, n;
  fscanf(inputFile, "%d %d", &m, &n);

  long int *red = new long int[m * n];   /* red channel */
  long int *green = new long int[m * n]; /* green channel */
  long int *blue = new long int[m * n];  /* blue channel */
  long int *T1 = new long int[m * n];
  long int *T2 = new long int[m * n];

  int num;
  int channel = 0;
  long int counter = 0;
  while (fscanf(inputFile, "%d", &num) != EOF) {
    switch (channel) {
    case 0:
      red[counter] = num;
      break;

    case 1:
      green[counter] = num;
      break;

    case 2:
      blue[counter] = num;
      break;
    }
    counter++;
    if (counter % (m * n) == 0) {
      channel++;
      counter = 0;
    }
  }

  /**
   *
   * DO NOT CHANGE ANYTHING ABOVE THIS LINE
   *
   */

  long int *d_red;
  long int *d_green;
  long int *d_blue;
  long int *d_T1;
  long int *d_T2;

  // allocating memory
  hipMalloc(&d_red, m * n * sizeof(long int));
  hipMalloc(&d_green, m * n * sizeof(long int));
  hipMalloc(&d_blue, m * n * sizeof(long int));
  hipMalloc(&d_T1, m * n * sizeof(long int));
  hipMalloc(&d_T2, m * n * sizeof(long int));

  // we need copy the matrices to the device matrices
  hipMemcpy(d_red, red, m * n * sizeof(long int), hipMemcpyHostToDevice);
  hipMemcpy(d_green, green, m * n * sizeof(long int), hipMemcpyHostToDevice);
  hipMemcpy(d_blue, blue, m * n * sizeof(long int), hipMemcpyHostToDevice);

  dim3 threadsPerBlock(1024, 1, 1);
  dim3 blocksPerGrid(ceil(m * n / 1024.0), 1, 1);

  auto start = std::chrono::high_resolution_clock::now();
  // write function
  // Note that d_T1 has the resultant matrix of Inverted Gray Scale matrix
  // Note that d_T2 has the resultant matrix of Thomas Transformation matrix
  CalculateInvertedGrayScale<<<blocksPerGrid, threadsPerBlock>>>(
      d_red, d_green, d_blue, d_T1, m, n);
  CalculateThomasTransformation<<<blocksPerGrid, threadsPerBlock>>>(
      d_red, d_green, d_blue, d_T2, m, n);
  hipDeviceSynchronize();

  hipMemcpy(T1, d_T1, m * n * sizeof(long int), hipMemcpyDeviceToHost);
  hipMemcpy(T2, d_T2, m * n * sizeof(long int), hipMemcpyDeviceToHost);
  hipFree(d_T1);
  hipFree(d_T2);
  hipFree(d_red);
  hipFree(d_green);
  hipFree(d_blue);
  auto end = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed1 = end - start;

  // Make sure your final output from the device is stored in d_T1.

  /**
   *
   * DO NOT CHANGE ANYTHING BELOW THIS LINE
   *
   */

  std::ofstream file("cuda.out");
  if (file.is_open()) {
    for (long int i = 0; i < m; i++) {
      for (long int j = 0; j < n; j++) {
        file << T1[i * n + j] << " ";
      }
      file << "\n";
    }
    for (long int i = 0; i < m; i++) {
      for (long int j = 0; j < n; j++) {
        file << T2[i * n + j] << " ";
      }
      file << "\n";
    }
    file.close();
  } else {
    std::cout << "Unable to open file";
  }

  std::ofstream file2("cuda_timing.out");
  if (file2.is_open()) {
    file2 << elapsed1.count() << "\n";
    file2.close();
  } else {
    std::cout << "Unable to open file";
  }

  return 0;
}
